#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template<typename scalar_t>
//__global__ void hello_cuda_kernel(const scalar_t * __restrict__ input)
__global__ void hello_cuda_kernel(
  const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> input) {
  if(threadIdx.x == 0) {
    printf("hello from tensor:\n");
    for(int i=0; i<input.size(0); i++)
      printf("%d, ", input[i]);
    printf("finish!\n");
  }
}

void hello_cuda(torch::Tensor input) {
   const int threads=32;
   const int blocks=1;

   AT_DISPATCH_ALL_TYPES(input.type(), "hello_cuda", 
     ([&] {
        hello_cuda_kernel<scalar_t><<<blocks, threads>>>
	//(input.data<scalar_t>());
	(input.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>());
   }));
}
