#include "hip/hip_runtime.h"
// #include <ATen/ATen.h>
// #include <ATen/core/Tensor.h>
// #include <ATen/Utils.h>
#include <c10/cuda/CUDAException.h>
#include <torch/extension.h>
#include <torch/library.h>

using namespace at;


int64_t integer_round(int64_t num, int64_t denom){
  return (num + denom - 1) / denom;
}


template<class T>
__global__ void add_one_kernel(const T *const input, T *const output, const int64_t N){
  // Grid-strided loop
  for(int i=blockDim.x*blockIdx.x+threadIdx.x;i<N;i+=blockDim.x*gridDim.x){
    output[i] = input[i] + 1;
  }
}


Tensor add_one(const Tensor &input){
  auto output = torch::zeros_like(input);

  AT_DISPATCH_ALL_TYPES(
    input.scalar_type(), "add_one_cuda", [&](){
      const auto block_size = 128;
      const auto num_blocks = std::min(65535L, integer_round(input.numel(), block_size));
      add_one_kernel<<<num_blocks, block_size>>>(
        input.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        input.numel()
      );
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  );

  return output;
}


TORCH_LIBRARY(pytorch_cmake_example, m) {
  m.def("add_one(Tensor input) -> Tensor");
  m.impl("add_one", c10::DispatchKey::CUDA, TORCH_FN(add_one));
}
